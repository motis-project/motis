#include "motis/raptor/gpu/gpu_timetable.cuh"

#include "utl/concat.h"

#include "motis/raptor/gpu/cuda_util.h"
#include "motis/raptor/raptor_util.h"

namespace motis::raptor {

template <typename T>
inline void copy_vector_to_device(std::vector<T> const& vec, T** ptr) {
  static_assert(std::is_trivially_copyable_v<T>);

  const auto size_in_bytes = vec_size_bytes(vec);
  hipMalloc(ptr, size_in_bytes);
  cuda_check();
  hipMemcpy(*ptr, vec.data(), size_in_bytes, hipMemcpyHostToDevice);
  cuda_check();
}

std::unique_ptr<host_gpu_timetable> get_host_gpu_timetable(
    raptor_timetable const& tt) {
  auto h_gtt = std::make_unique<host_gpu_timetable>();

  // Copy the members, which are identical on CPU and GPU
  h_gtt->stops_ = tt.stops_;
  h_gtt->routes_ = tt.routes_;

  h_gtt->route_stops_ = tt.route_stops_;
  h_gtt->stop_routes_ = tt.stop_routes_;

  // Create GPU footpaths, with from and to station
  h_gtt->footpaths_.resize(tt.footpath_count());
  for (stop_id s_id = 0; s_id < tt.stop_count(); ++s_id) {
    auto const& stop = tt.stops_[s_id];
    auto const& next_stop = tt.stops_[s_id + 1];

    for (auto foot_idx = stop.index_to_transfers_;
         foot_idx < next_stop.index_to_transfers_; ++foot_idx) {
      auto const& f = tt.footpaths_[foot_idx];
      h_gtt->footpaths_[foot_idx].from_ = s_id;
      h_gtt->footpaths_[foot_idx].to_ = f.to_;
      h_gtt->footpaths_[foot_idx].duration_ = f.duration_;
    }
  }

  // Create split stop times arrays
  h_gtt->stop_arrivals_.reserve(tt.stop_times_.size());
  h_gtt->stop_departures_.reserve(tt.stop_times_.size());
  for (auto const stop_time : tt.stop_times_) {
    h_gtt->stop_arrivals_.push_back(stop_time.arrival_);
    h_gtt->stop_departures_.push_back(stop_time.departure_);
  }

  return h_gtt;
}

std::unique_ptr<device_gpu_timetable> get_device_gpu_timetable(
    host_gpu_timetable const& h_gtt) {
  auto d_gtt = std::make_unique<device_gpu_timetable>();

  copy_vector_to_device(h_gtt.stops_, &(d_gtt->stops_));
  copy_vector_to_device(h_gtt.routes_, &(d_gtt->routes_));

  copy_vector_to_device(h_gtt.footpaths_, &(d_gtt->footpaths_));

  copy_vector_to_device(h_gtt.stop_times_, &(d_gtt->stop_times_));

  copy_vector_to_device(h_gtt.stop_arrivals_, &(d_gtt->stop_arrivals_));
  copy_vector_to_device(h_gtt.stop_departures_, &(d_gtt->stop_departures_));

  copy_vector_to_device(h_gtt.route_stops_, &(d_gtt->route_stops_));
  copy_vector_to_device(h_gtt.stop_routes_, &(d_gtt->stop_routes_));

  d_gtt->stop_count_ = h_gtt.stop_count();
  d_gtt->route_count_ = h_gtt.route_count();
  d_gtt->footpath_count_ = h_gtt.footpaths_.size();

  copy_vector_to_device(h_gtt.transfer_times_, &(d_gtt->transfer_times_));

  return d_gtt;
}

void destroy_device_gpu_timetable(device_gpu_timetable& d_gtt) {
  hipFree(d_gtt.stops_);
  hipFree(d_gtt.routes_);
  hipFree(d_gtt.footpaths_);
  hipFree(d_gtt.transfer_times_);
  hipFree(d_gtt.stop_times_);
  hipFree(d_gtt.stop_arrivals_);
  hipFree(d_gtt.stop_departures_);
  hipFree(d_gtt.route_stops_);
  hipFree(d_gtt.stop_routes_);
}

}  // namespace motis::raptor
